#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>

#include <ingot/ingot.h>
using namespace ingot;

struct integration_step {

    ode::CR3BP cr3bp{0.04};

    template<typename T>
    __device__ void operator()(T arg) {

        auto& t = thrust::get<0>(arg); // time
        auto& h = thrust::get<1>(arg); // timestep
        auto x = thrust::get<2>(arg);  //

        constexpr auto N = decltype(x)::SizeAtCompileTime;
        static_assert(N != Eigen::Dynamic,
                      "ODE state-vector must be statically sized!");

        StackArray<double, 6> x_old = thrust::get<2>(arg);

        // Eigen::Array<double, 6, 1> x_new = RK4{}(cr3bp, t, h, x_old);
        Eigen::Array<double, 6, 1> x_new = RKF78{}(cr3bp, t, h, x_old);

        t += h;
        x = x_new;

        if (threadIdx.x == 0 and blockIdx.x == 0) {
            const auto s = x_new;
            printf("%g %g %g %g %g %g\n", s[0], s[1], s[2], s[3], s[4], s[5]);
        }
    }
};

struct initializer {
    __device__ auto operator()(int i) const {

        const auto vmag = 0.1;
        const auto theta = i * M_PI;

        const auto vx = vmag * cos(theta);
        const auto vy = vmag * sin(theta);

        Eigen::Array<double, 6, 1> ret;
        ret << -.5, 0, 0, vx, vy, 0;

        return ret;
    }
};

int main() {
    using T = double;

    // Allocate initial buffer
    const int nparticles = 1024;
    thrust::device_vector<T> x{nparticles};

    ColIter<T, 6> ci{x.data().get(), nparticles};

    auto idx = thrust::counting_iterator<int>{0};

    thrust::transform(idx, idx + nparticles, ci, initializer{});

    thrust::device_vector<T> t{nparticles};
    thrust::fill(t.begin(), t.end(), 0);
    thrust::device_vector<T> h{nparticles};
    // thrust::fill(h.begin(), h.end(), std::numeric_limits<T>::epsilon());
    thrust::fill(h.begin(), h.end(), .01);

    auto zp = zip_tuple_iters(t.begin(), h.begin(), ci);

    for (int i = 0; i < 500; i++)
        thrust::for_each(zp, zp + nparticles, integration_step{});
}
