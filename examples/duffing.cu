#include "hip/hip_runtime.h"
#include <ingot/ingot.h>
#include <ingot/integrate.h>
#include <ingot/integrator/adaptive.h>
#include <ingot/method/RKF78.h>

namespace ingot {
namespace ode {

// x'' + delta x' + alpha x + beta x^3 = gamma cos(omega t)

class Duffing {

public:
    double delta;
    double alpha;
    double beta;
    double gamma;
    double omega;

    template<typename T>
    CUDA_HOSTDEV constexpr auto operator()(const double t, Eigen::Array<T, 2, 1> const& u) const {

        Eigen::Array<T, 2, 1> up; // u-prime

        const double x = u[0];

        up[0] = u[1];
        up[1] = gamma * cos(omega * t) - delta * u[1] - alpha * x - beta * x*x*x;

        return up;
    }
};

} // namespace ode
} // namespace ingot

using namespace ingot;

// Did the sign of the x-component change?
template<typename T, int N>
struct XVal {
    CUDA_HOSTDEV
    T operator()(double t, double h, ColVal<T, N> const& u) const {
        return u[0];
    }
};

CUDA_HOST static double rand_gen() { return 0.1 + double(rand()) / RAND_MAX / 2; }

int main() {

    /*
     * Allocate and initialize particles
     */
    int size = 4;
    Ensemble<double, 2> ensemble{size};
    thrust::fill(ensemble.t.begin(), ensemble.t.end(), 0);
    thrust::fill(ensemble.h.begin(), ensemble.h.end(), .1);
    {
        srand(392);
        thrust::host_vector<double> hv{2 * size};
        auto u0 = hv.begin();
        thrust::generate_n(u0, size, rand_gen);
        thrust::fill(u0 + size, hv.end(), 0);
        ensemble.u.data = hv;
    }

    double tmax = 10;
    auto i = integrator::make_adaptive(method::RKF78{}, 1e-8);
    i.h_max = M_PI / 2;

    const double delta = 0.02;
    const double alpha = 1;
    const double beta = 5;
    const double gamma = 8;
    const double omega = 0.5;

    auto duffing_ode = ode::Duffing{delta, alpha, beta, gamma, omega};

    const auto sols = integrate_dense(i, duffing_ode, ensemble, tmax);
    int idx = 0;
    for (const auto& isols : sols) {
        for (const auto& s : isols) {
            printf("%d : %g: %g, %g\n", idx, s.t, s.u[0], s.u[1]);
        }
        idx++;
    }

    printf("\nNumber of solutions: %d\n", sols.size());
}
