#include "hip/hip_runtime.h"
#include <doctest/doctest.h>
#include <ingot/ingot.h>
#include <ingot/integrate.h>
#include <ingot/integrator/adaptive.h>
#include <ingot/method/RKF78.h>

namespace ingot {
namespace ode {

class SHO : public HostDevTimeInvariantODE<SHO> {

    using base_t = HostDevTimeInvariantODE<SHO>;

    double k;

public:
    CUDA_HOSTDEV
    SHO(double k) : k{k} {}

    using base_t::operator();

    template<typename T>
    CUDA_HOSTDEV void operator()(Eigen::Array<T, 2, 1>& up, // u-prime
                                 Eigen::Array<T, 2, 1> const& u) const {

        up[0] = u[1];
        up[1] = -k * u[0];
    }
};

} // namespace ode
} // namespace ingot

using namespace ingot;

// Did the sign of the x-component change?
template<typename T, int N>
struct XVal {
    CUDA_HOSTDEV
    T operator()(double t, double h, ColVal<T, N> const& u) const {
        return u[0];
    }
};

CUDA_HOST static double rand_gen() { return 0.1 + double(rand()) / RAND_MAX / 2; }

TEST_CASE("SimpleHarmonicOscillator, EnsembleAdaptive") {

    /*
     * Allocate and initialize particles
     */
    int size = 4;
    Ensemble<double, 2> ensemble{1};
    thrust::fill(ensemble.t.begin(), ensemble.t.end(), 0);
    thrust::fill(ensemble.h.begin(), ensemble.h.end(), .1);
    {
        thrust::host_vector<double> hv{2 * size};
        hv[0] = 1;
        hv[1] = 0;
        ensemble.u.data = hv;
    }

    double tmax = 10;
    auto i = integrator::make_adaptive(method::RKF78{}, 1e-8);
    i.h_max = M_PI / 2;

    const auto sols = integrate_steps(i, ode::SHO{1}, ensemble, tmax / 0.1,
                                      XVal<double, 2>{});

    double tol = 1e-6;

    double current_sol = M_PI / 2;
    for (const auto& s : sols) {

        // check current time
        CHECK(s.t - current_sol < tol);
        current_sol += M_PI;

        // check position
        CHECK(s.u[0] < tol);

        // check velocity
        CHECK(fabs(s.u[1]) - 1 < tol);
    }

    // make sure we actually checked some solutions :)
    CHECK(sols.size() > 10);
}
