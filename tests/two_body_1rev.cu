#include "hip/hip_runtime.h"
#include <gtest/gtest.h>

#include <ingot/ingot.h>
using namespace ingot;

TEST(TwoBody, CircularOrbit) {

    double sv0[6]{1, 0, 0, 0, 1, 0};

    double tspan[2] = {0., 2 * M_PI};

    auto prob = ODEProblem(ode::TwoBody{}, sv0, tspan);

    SolveArgs args;
    args.h0 = 0.1;
    auto sols = solve(prob, RK4{}, args);

    StackArray<double, 6> first{sv0};
    auto last = sols.back().u;

    EXPECT_NEAR((first - last).norm(), 0, 1e-8);
}
