#include "hip/hip_runtime.h"
#include <gtest/gtest.h>

#include <ingot/ingot.h>
#include <ingot/integrator/adaptive.h>
#include <ingot/method/all.h>
#include <ingot/ode/TwoBody.h>
using namespace ingot;

TEST(TwoBodyCircularOrbit, CpuFixed) {

    double sv0[6]{1, 0, 0, 0, 1, 0};

    double tspan[2] = {0., 2 * M_PI};

    auto prob = ODEProblem(ode::TwoBody{}, sv0, tspan);

    SolveArgs args;
    args.h0 = 0.1;
    StackArray<double, 6> first{sv0};

    auto sols = solve(prob, method::RK4{}, args);
    auto last = sols.back().u;
    auto diff = (first - last).norm();
    EXPECT_GT(diff, std::numeric_limits<double>::epsilon());
    EXPECT_LT(diff, 3.3e-5);

    sols = solve(prob, method::DoPri45{}, args);
    last = sols.back().u;
    diff = (first - last).norm();
    EXPECT_GT(diff, std::numeric_limits<double>::epsilon());
    EXPECT_LT(diff, 1.1e-7);

    sols = solve(prob, method::Tsit5{}, args);
    last = sols.back().u;
    diff = (first - last).norm();
    EXPECT_GT(diff, std::numeric_limits<double>::epsilon());
    EXPECT_LT(diff, 1.1e-7);

    sols = solve(prob, method::RKF78{}, args);
    last = sols.back().u;
    diff = (first - last).norm();
    EXPECT_GT(diff, std::numeric_limits<double>::epsilon());
    EXPECT_LT(diff, 7e-12);
}

struct probfunc {
    CUDA_DEV auto operator()(int i, StackArray<double, 6> y) const {
        return y;
    }
};

TEST(TwoBodyCircularOrbit, EnsembleFixed) {

    double sv0[6]{1, 0, 0, 0, 1, 0};

    double tspan[2] = {0., 2 * M_PI};

    auto prob = ODEProblem(ode::TwoBody{}, sv0, tspan);
    auto eprob = EnsembleProblem(prob, probfunc{});

    SolveArgs args;
    args.h0 = 0.1;
    StackArray<double, 6> first{sv0};

    {
        const auto i = integrator::make_fixed(method::RK4{});
        const auto sols = solve(eprob, i, 1, args);
        const auto last = sols.back().u;
        const auto diff = (first - last).norm();
        EXPECT_GT(diff, std::numeric_limits<double>::epsilon());
        EXPECT_LT(diff, 3.3e-5);
    }

    {
        const auto i = integrator::make_fixed(method::DoPri45{});
        const auto sols = solve(eprob, i, 1, args);
        const auto last = sols.back().u;
        const auto diff = (first - last).norm();
        EXPECT_GT(diff, std::numeric_limits<double>::epsilon());
        EXPECT_LT(diff, 1.1e-7);
    }

    {
        const auto i = integrator::make_fixed(method::Tsit5{});
        const auto sols = solve(eprob, i, 1, args);
        const auto last = sols.back().u;
        const auto diff = (first - last).norm();
        EXPECT_GT(diff, std::numeric_limits<double>::epsilon());
        EXPECT_LT(diff, 1.1e-7);
    }

    {
        const auto i = integrator::make_fixed(method::RKF78{});
        const auto sols = solve(eprob, i, 1, args);
        const auto last = sols.back().u;
        const auto diff = (first - last).norm();
        EXPECT_GT(diff, std::numeric_limits<double>::epsilon());
        EXPECT_LT(diff, 7e-12);
    }
}

TEST(TwoBodyCircularOrbit, EnsembleAdaptive) {

    double sv0[6]{1, 0, 0, 0, 1, 0};

    double tspan[2] = {0., 2 * M_PI};

    auto prob = ODEProblem(ode::TwoBody{}, sv0, tspan);
    auto eprob = EnsembleProblem(prob, probfunc{});

    SolveArgs args;
    args.h0 = 0.1;
    StackArray<double, 6> first{sv0};

    {
        const auto i = integrator::make_adaptive(method::DoPri45{}, 1e-8);
        const auto sols = solve(eprob, i, 1, args);
        const auto last = sols.back().u;
        const auto diff = (first - last).norm();
        EXPECT_GT(diff, std::numeric_limits<double>::epsilon());
        EXPECT_LT(diff, 1e-6);
    }

    {
        const auto i = integrator::make_adaptive(method::Tsit5{}, 1e-2);
        const auto sols = solve(eprob, i, 1, args);
        const auto last = sols.back().u;
        const auto diff = (first - last).norm();
        EXPECT_GT(diff, std::numeric_limits<double>::epsilon());
        EXPECT_LT(diff, 1e-6);
    }

    {
        const auto i = integrator::make_adaptive(method::RKF78{}, 1e-8);
        const auto sols = solve(eprob, i, 1, args);
        const auto last = sols.back().u;
        const auto diff = (first - last).norm();
        EXPECT_GT(diff, std::numeric_limits<double>::epsilon());
        EXPECT_LT(diff, 1e-6);
    }
}
