#include "hip/hip_runtime.h"
#include <ingot/ingot.h>
using namespace ingot;

#include <gtest/gtest.h>

TEST(TwoBody, CircularOrbit) {

    double sv0[6]{1, 0, 0, 0, 1, 0};

    double tspan[2] = {0., 2 * M_PI};

    auto prob = ODEProblem(ode::TwoBody{1}, sv0, tspan);

    auto sols = solve(prob, RK4{});

    StackArray<double, 6> first{sv0};
    auto last = sols.back().u;

    EXPECT_NEAR((first - last).norm(), 0, 1e-5);
}
