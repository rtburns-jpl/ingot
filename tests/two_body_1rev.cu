#include "hip/hip_runtime.h"
#include <gtest/gtest.h>

#include <ingot/ingot.h>
using namespace ingot;

TEST(TwoBody, CircularOrbit) {

    double sv0[6]{1, 0, 0, 0, 1, 0};

    double tspan[2] = {0., 2 * M_PI};

    auto prob = ODEProblem(ode::TwoBody{}, sv0, tspan);

    SolveArgs args;
    args.h0 = 0.1;
    auto sols = solve(prob, method::RK4{}, args);

    StackArray<double, 6> first{sv0};
    auto last = sols.back().u;

    EXPECT_NEAR((first - last).norm(), 0, 1e-8);
}

struct probfunc {
    CUDA_DEV auto operator()(int i, StackArray<double, 6> y) const {
        return y;
    }
};

TEST(TwoBody, CircularOrbitEnsemble) {

    double sv0[6]{1, 0, 0, 0, 1, 0};

    double tspan[2] = {0., 2 * M_PI};

    auto prob = ODEProblem(ode::TwoBody{}, sv0, tspan);
    auto eprob = EnsembleProblem(prob, probfunc{});

    SolveArgs args;
    args.h0 = 0.1;
    solve(eprob, method::RK4{}, 100, args);

    /*
    StackArray<double, 6> first{sv0};
    auto last = sols.back().u;

    EXPECT_NEAR((first - last).norm(), 0, 1e-8);
    */
}
