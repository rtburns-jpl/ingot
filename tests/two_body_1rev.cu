#include "hip/hip_runtime.h"
#include <gtest/gtest.h>

#include <ingot/ingot.h>
using namespace ingot;

TEST(TwoBody, CircularOrbit) {

    double sv0[6]{1, 0, 0, 0, 1, 0};

    double tspan[2] = {0., 2 * M_PI};

    auto prob = ODEProblem(ode::TwoBody{}, sv0, tspan);

    SolveArgs args;
    args.h0 = 0.1;
    StackArray<double, 6> first{sv0};

    auto sols = solve(prob, method::RK4{}, args);
    auto last = sols.back().u;
    EXPECT_LT((first - last).norm(), 1.1e-9);

    sols = solve(prob, method::DoPri45{}, args);
    last = sols.back().u;
    EXPECT_LT((first - last).norm(), 6e-15);

    sols = solve(prob, method::Tsit5{}, args);
    last = sols.back().u;
    EXPECT_LT((first - last).norm(), 1.1e-14);

    sols = solve(prob, method::RKF78{}, args);
    last = sols.back().u;
    EXPECT_LT((first - last).norm(), 5e-23);
}

struct probfunc {
    CUDA_DEV auto operator()(int i, StackArray<double, 6> y) const {
        return y;
    }
};

TEST(TwoBody, CircularOrbitEnsemble) {

    double sv0[6]{1, 0, 0, 0, 1, 0};

    double tspan[2] = {0., 2 * M_PI};

    auto prob = ODEProblem(ode::TwoBody{}, sv0, tspan);
    auto eprob = EnsembleProblem(prob, probfunc{});

    SolveArgs args;
    args.h0 = 0.1;
    auto sols = solve(eprob, method::RK4{}, 100, args);

    /*
    StackArray<double, 6> first{sv0};
    auto last = sols.back().u;

    EXPECT_NEAR((first - last).norm(), 0, 1e-8);
    */
}
