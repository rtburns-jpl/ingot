#include <ingot/ingot.h>
#include <ingot/ode/CR3BP.h>
#include <ingot/integrator/adaptive.h>
#include <ingot/integrate.h>
#include <ingot/method/RKF78.h>

#include <pybind11/pybind11.h>
#include <pybind11/eigen.h>

namespace py = pybind11;
using namespace ingot;

// Did the sign of the y-component change?
template<typename T, int N>
struct YVal {
    CUDA_HOSTDEV
    T operator()(double t, double h, ColVal<T, N> const& u) const {
        return u[1];
    }
};

#define cuCheck(x) cuCheckImpl((x), __PRETTY_FUNCTION__, __LINE__)

void cuCheckImpl(hipError_t x, std::string func, int line) {
    if (x != hipSuccess) {
        throw std::runtime_error("In function " + func +
                                 " line " + std::to_string(line));
    }
}

auto integrate_cr3bp_rkf78_steps(
        double mu,
        Eigen::Ref<Eigen::VectorXd> host_t,
        Eigen::Ref<Eigen::VectorXd> host_h,
        Eigen::Ref<Eigen::MatrixXd> host_u
        ) {

    /*
     * Allocate and initialize particles
     */
    const int size = host_t.size();

    if (host_h.size() != size) {
        throw std::invalid_argument("t.size() != h.size()");
    }
    if (host_u.cols() != size) {
        throw std::invalid_argument("t.size() != u.cols()");
    }
    if (host_u.rows() != 6) {
        throw std::invalid_argument("u.rows() != 6");
    }

    Ensemble<double, 6> ensemble{size};

#define HtoD hipMemcpyHostToDevice

    cuCheck(hipMemcpy(ensemble.t.data().get(), host_t.data(),
                size * sizeof(double), HtoD));
    cuCheck(hipMemcpy(ensemble.h.data().get(), host_t.data(),
                size * sizeof(double), HtoD));
    cuCheck(hipMemcpy2D(ensemble.u.data.data().get(), size * sizeof(double),
                         host_u.data(), size * sizeof(double),
                         size * sizeof(double), 6, HtoD));

    /*
     * Integrate with output function for fixed number of steps
     */
    const auto i = integrator::make_adaptive(method::RKF78{}, 1e-8);
    const auto sols = integrate_steps(i, ode::CR3BP{mu}, ensemble, 10000,
                                      YVal<double, 6>{});

    return sols;
}

PYBIND11_MODULE(PY_EXT_NAME, m) {

    using namespace ingot;

    py::class_<output<double, 6>>(m, "Output3D")
        .def_readonly("t", &output<double, 6>::t)
        .def_readonly("h", &output<double, 6>::h)
        .def_readonly("u", &output<double, 6>::u)
        ;

    m.def("integrate_cr3bp_rkf78_steps", integrate_cr3bp_rkf78_steps);
}
